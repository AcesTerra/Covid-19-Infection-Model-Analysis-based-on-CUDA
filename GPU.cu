#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <ctime>
#include <cmath>
#include <iostream>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

using namespace std;

// Agent structure with all properties
struct agent {
	double infectionProb;			// [0.02, 0.03]
	double externalInfectionProb;	// [0.02, 0.03]
	double mortalityProb;			// [0.007, 0.07]
	double mobilityProb;			// [0.3, 0.5]
	double shortMobilityProb;		// [0.7, 0.9]
	int incubationTime;				// [5, 6]
	int recoveryTime;				// 14
	int infectionStatus;			// Non infected (0), infected (1), quarantine (-1), deseaced (-2), cured (2)
	double x;						// [0, p]
	double y;						// [0, q]
};

// Simulation parameters
#define numberOfAgents 1024
const int maxSimulationDays = 30;
const int maxMovementsPerDay = 10;
#define maximumRadiusForLocalMovements 5
const float infectionLimitDistance = 1;
#define p 500
#define q 500

// Function to generate random int numbers with CUDA
__device__ int generateRandomIntCUDA(int gID) {
	hiprandState_t state;
	hiprand_init((unsigned long long)clock() + gID, 0, 0, &state);
	int result = hiprand(&state);
	return abs(result);
}

// Function to generate random float numbers with CUDA
__device__ double generateRandomFloatCUDA(float min, float max)
{
	int gID = blockIdx.x * blockDim.x + threadIdx.x;
	hiprandState state;
	hiprand_init((unsigned long long)clock() + gID, 0, 0, &state);

	double result = hiprand_uniform_double(&state) * (max - min) + min;
	return result;
}

// Function to check CUDA errors
__host__ void check_CUDA_error(const char* msj) {
	hipError_t error;
	hipDeviceSynchronize();
	error = hipGetLastError();
	if (error != hipSuccess) {
		printf("Error: %d %s (%s)", error, hipGetErrorString(error), msj);
	}
}

// Function to move a short distance an agent
__device__ float shortMovement(float pos) {
	float newPos = (2 * generateRandomFloatCUDA(0.0, 1.0) - 1) * maximumRadiusForLocalMovements + pos;
	if (newPos > 500)
		newPos = 500;
	if (newPos < 0)
		newPos = 0;
	return newPos;
}

// Function to move a long distance in X an agent
__device__ float longXMovement(float pos) {
	float newPos = p * generateRandomFloatCUDA(-1.0, 1.0) + pos;
	if (newPos > 500)
		newPos = 500;
	if (newPos < 0)
		newPos = 0;
	return newPos;
}

// Function to move a long distance in Y an agent
__device__ float longYMovement(float pos) {
	float newPos = q * generateRandomFloatCUDA(-1.0, 1.0) + pos;
	if (newPos > 500)
		newPos = 500;
	if (newPos < 0)
		newPos = 0;
	return newPos;
}

// Function to initalize agents properties
__global__ void initializeAgents(agent allAgents[]) {
	int gID = blockIdx.x * blockDim.x + threadIdx.x;
	allAgents[gID].infectionProb = generateRandomFloatCUDA(0.02, 0.03);
	allAgents[gID].externalInfectionProb = generateRandomFloatCUDA(0.02, 0.03);
	allAgents[gID].mortalityProb = generateRandomFloatCUDA(0.007, 0.07);
	allAgents[gID].mobilityProb = generateRandomFloatCUDA(0.3, 0.5);
	allAgents[gID].shortMobilityProb = generateRandomFloatCUDA(0.7, 0.9);
	allAgents[gID].incubationTime = generateRandomIntCUDA(gID) % 2 + 5;
	allAgents[gID].recoveryTime = 14;
	allAgents[gID].infectionStatus = 0;
	allAgents[gID].x = generateRandomFloatCUDA(0.0, (float)p);
	allAgents[gID].y = generateRandomFloatCUDA(0.0, (float)q);
}
// Function to show all agents properties
__host__ void showAgents(agent allAgents[]) {
	for (int i = 0; i < numberOfAgents; i++) {
		printf("Agent's no. %d probability of infection: %f\n", i + 1, allAgents[i].infectionProb);
		printf("Agent's no. %d external probability of infection: %f\n", i + 1, allAgents[i].externalInfectionProb);
		printf("Agent's no. %d probability of mortality: %f\n", i + 1, allAgents[i].mortalityProb);
		printf("Agent's no. %d probability of mobility: %f\n", i + 1, allAgents[i].mobilityProb);
		printf("Agent's no. %d probability of short mobility: %f\n", i + 1, allAgents[i].shortMobilityProb);
		printf("Agent's no. %d incubation time: %d\n", i + 1, allAgents[i].incubationTime);
		printf("Agent's no. %d recovery time: %d\n", i + 1, allAgents[i].recoveryTime);
		printf("Agent's no. %d infection status: %d\n", i + 1, allAgents[i].infectionStatus);
		printf("Agent's no. %d x position: %f\n", i + 1, allAgents[i].x);
		printf("Agent's no. %d y position: %f\n\n", i + 1, allAgents[i].y);
	}
}

// Rule 1: Infection
__global__ void ruleOne(agent agents[], int historyCounter[]) {
	int gID = blockIdx.x * blockDim.x + threadIdx.x;
	for (int j = 0; j < numberOfAgents; j++) {
		double distance = sqrt(pow(agents[gID].x - agents[j].x, 2.0) + pow(agents[gID].y - agents[j].y, 2.0));
		if (distance <= 1.0 && agents[j].infectionStatus == 1 && agents[gID].infectionStatus == 0 && gID != j) {
			float infection = generateRandomFloatCUDA(0.0, 1.0);
			if (infection <= agents[gID].infectionProb) {
				agents[gID].infectionStatus = 1;
				historyCounter[gID] = 1;
			}
		}
	}
}
// Rule 2: Mobility
__global__ void ruleTwo(agent agents[]) {
	int gID = blockIdx.x * blockDim.x + threadIdx.x;
	float movProb = generateRandomFloatCUDA(0.0, 1.0);
	if (movProb <= agents[gID].mobilityProb && (agents[gID].infectionStatus == 0 || agents[gID].infectionStatus == 1)) {
		float shortMovProb = generateRandomFloatCUDA(0.0, 1.0);
		float newXPos, newYPos;
		if (shortMovProb <= agents[gID].shortMobilityProb) {
			newXPos = shortMovement(agents[gID].x);
			newYPos = shortMovement(agents[gID].y);
			agents[gID].x = newXPos;
			agents[gID].y = newYPos;
		}
		else {
			newXPos = longXMovement(agents[gID].x);
			newYPos = longYMovement(agents[gID].y);
			agents[gID].x = newXPos;
			agents[gID].y = newYPos;
		}
	}
}

// Rule 3: External infection
__global__ void ruleThree(agent agents[], int historyCounter[]) {
	int gID = blockIdx.x * blockDim.x + threadIdx.x;
	float infectionExternal = generateRandomFloatCUDA(0.0, 1.0);
	if (infectionExternal <= agents[gID].externalInfectionProb && agents[gID].infectionStatus == 0) {
		agents[gID].infectionStatus = 1;
		historyCounter[gID] = 1;
	}
}

// Rule 4: Incucation time, symptoms, quarantine and recovery time
__global__ void ruleFour(agent agents[], int historyCounter[]) {
	int gID = blockIdx.x * blockDim.x + threadIdx.x;
	if (agents[gID].infectionStatus == -1 && agents[gID].recoveryTime > 0) {
		agents[gID].recoveryTime = agents[gID].recoveryTime - 1;
	}
	if (agents[gID].infectionStatus == 1 && agents[gID].incubationTime > 0) {
		agents[gID].incubationTime = agents[gID].incubationTime - 1;
	}
	if (agents[gID].infectionStatus == 1 && agents[gID].incubationTime == 0) {
		agents[gID].infectionStatus = -1;
	}
	if (agents[gID].infectionStatus == -1 && agents[gID].recoveryTime == 0) {
		agents[gID].infectionStatus = 2;
		historyCounter[gID] = 1;
	}
}
// Rule 5: Fatal cases
__global__ void ruleFive(agent agents[], int historyCounter[]) {
	int gID = blockIdx.x * blockDim.x + threadIdx.x;
	float fatal = generateRandomFloatCUDA(0.0, 1.0);
	if (fatal <= agents[gID].mortalityProb && agents[gID].infectionStatus == -1) {
		agents[gID].infectionStatus = -2;
		historyCounter[gID] = 1;
	}
}

//Function to sum the history of specific day
__global__ void sumHistory(int historyCounter[]) {
	int gID = blockIdx.x * blockDim.x + threadIdx.x;
	__syncthreads();
	int jump = numberOfAgents / 2;
	while (jump) {
		if (gID < jump) {
			historyCounter[gID] = historyCounter[gID] + historyCounter[gID + jump];
		}
		__syncthreads();
		jump = jump / 2;
	}
}

// Function to update the history of pandemic
__global__ void updateHistory(int day, int historyCounter[], int historyToUpdate[]) {
	historyToUpdate[day] = historyCounter[0];
}

// Function to initialize device counters
__global__ void initializeHistory(int counterControl_dev[], int historyCounter_dev[]) {
	int gID = blockIdx.x * blockDim.x + threadIdx.x;
	counterControl_dev[gID] = 0;
	historyCounter_dev[gID] = 0;
}

int main() {
	/*
	*************************************************************
	******************** Initalization phase ********************
	*************************************************************
	*/

	hipEvent_t start_GPU;
	hipEvent_t end_GPU;
	hipEventCreate(&start_GPU);
	hipEventCreate(&end_GPU);
	hipEventRecord(start_GPU, 0);

	int allInfectionsCounter_host = 0;
	int* infectionHistory_host;
	infectionHistory_host = (int*)malloc(maxSimulationDays * sizeof(int));
	int allRecoveryCounter_host = 0;
	int* recoveryHistory_host;
	recoveryHistory_host = (int*)malloc(maxSimulationDays * sizeof(int));
	int allFatalCounter_host = 0;
	int* fatalHistory_host;
	fatalHistory_host = (int*)malloc(maxSimulationDays * sizeof(int));

	int* allInfectionsCounter_dev;
	hipMalloc((void**)&allInfectionsCounter_dev, sizeof(int));
	int* infectionHistory_dev;
	hipMalloc((void**)&infectionHistory_dev, maxSimulationDays * sizeof(int));
	int* allRecoveryCounter_dev;
	hipMalloc((void**)&allRecoveryCounter_dev, sizeof(int));
	int* recoveryHistory_dev;
	hipMalloc((void**)&recoveryHistory_dev, maxSimulationDays * sizeof(int));
	int* allFatalCounter_dev;
	hipMalloc((void**)&allFatalCounter_dev, sizeof(int));
	int* fatalHistory_dev;
	hipMalloc((void**)&fatalHistory_dev, maxSimulationDays * sizeof(int));

	agent* allAgents_dev;
	hipMalloc((void**)&allAgents_dev, numberOfAgents * sizeof(agent));

	int* counterControl_dev, * historyCounter_dev;
	hipMalloc((void**)&counterControl_dev, numberOfAgents * sizeof(int));
	hipMalloc((void**)&historyCounter_dev, numberOfAgents * sizeof(int));

	dim3 block(32);
	dim3 grid(32);

	initializeAgents << <grid, block >> > (allAgents_dev);
	initializeHistory << <grid, block >> > (counterControl_dev, historyCounter_dev);
	hipDeviceSynchronize();
	check_CUDA_error("Error en kernel");

	printf("---------------------Simulation parameters---------------------\n");
	printf("\nNumber of agents: %d\n", numberOfAgents);
	printf("Simulation days: %d\n", maxSimulationDays);
	printf("Max movements per day: %d\n", maxMovementsPerDay);
	printf("Maximum radius for local movements: %d\n", maximumRadiusForLocalMovements);
	printf("Infection limit distance: %f\n", infectionLimitDistance);
	printf("P: %d\n", p);
	printf("Q: %d\n", q);
	printf("\n--------------------Initializing simulation--------------------\n");

	/*
	*************************************************************
	********************** Operation phase **********************
	*************************************************************
	*/

	for (int day = 0; day < maxSimulationDays; day++) {
		for (int mov = 0; mov < maxMovementsPerDay; mov++) {
			ruleOne << <grid, block >> > (allAgents_dev, historyCounter_dev);
			hipDeviceSynchronize();
			check_CUDA_error("Error en kernel");
			ruleTwo << <grid, block >> > (allAgents_dev);
			hipDeviceSynchronize();
			check_CUDA_error("Error en kernel");
		}
		ruleThree << <grid, block >> > (allAgents_dev, historyCounter_dev);
		sumHistory << <grid, block >> > (historyCounter_dev);
		updateHistory << <1, 1 >> > (day, historyCounter_dev, infectionHistory_dev);
		hipDeviceSynchronize();
		hipMemcpy(historyCounter_dev, counterControl_dev, numberOfAgents * sizeof(int), hipMemcpyDeviceToDevice);
		check_CUDA_error("Error en kernel");
		ruleFour << <grid, block >> > (allAgents_dev, historyCounter_dev);
		sumHistory << <grid, block >> > (historyCounter_dev);
		updateHistory << <1, 1 >> > (day, historyCounter_dev, recoveryHistory_dev);
		hipDeviceSynchronize();
		hipMemcpy(historyCounter_dev, counterControl_dev, numberOfAgents * sizeof(int), hipMemcpyDeviceToDevice);
		check_CUDA_error("Error en kernel");
		ruleFive << <grid, block >> > (allAgents_dev, historyCounter_dev);
		sumHistory << <grid, block >> > (historyCounter_dev);
		updateHistory << <1, 1 >> > (day, historyCounter_dev, fatalHistory_dev);
		hipDeviceSynchronize();
		hipMemcpy(historyCounter_dev, counterControl_dev, numberOfAgents * sizeof(int), hipMemcpyDeviceToDevice);
		check_CUDA_error("Error en kernel");
	}

	/*
	*************************************************************
	************************ Show results ***********************
	*************************************************************
	*/
	printf("\n---------------------Simulation terminated---------------------\n");

	hipMemcpy(infectionHistory_host, infectionHistory_dev, maxSimulationDays * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(recoveryHistory_host, recoveryHistory_dev, maxSimulationDays * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(fatalHistory_host, fatalHistory_dev, maxSimulationDays * sizeof(int), hipMemcpyDeviceToHost);
	check_CUDA_error("Error hipMemcpy");

	for (int i = 0; i < maxSimulationDays; i++) {
		allInfectionsCounter_host += infectionHistory_host[i];
		allRecoveryCounter_host += recoveryHistory_host[i];
		allFatalCounter_host += fatalHistory_host[i];
	}

	int zeroDayInfected = 0, halfPopulationInfected = 0, allPopulationInfected = 0;
	int zeroDayRecovered = 0, halfAgentsRecovered = 0, allAgentsRecovered = 0;
	int zeroDayFatal = 0, halfAgentsFatal = 0, allAgentsFatal = 0;
	int halfPopulationInfectedDay = 0, allPopulationInfectedDay = 0;
	int halfAgentsRecoveredDay = 0, allAgentsRecoveredDay = 0;
	int halfAgentsFatalDay = 0, allAgentsFatalDay = 0;
	bool zeroDayInfectedFlag = false, halfPopulationInfectedFlag = false;
	bool zeroDayRecoveredFlag = false, halfAgentsRecoveredFlag = false;
	bool zeroDayFatalFlag = false, halfAgentsFatalFlag = false;

	printf("\nTotal infected cases: %d\n", allInfectionsCounter_host);
	printf("Infection history: ");
	for (int i = 0; i < maxSimulationDays; i++) {
		printf("%d ", infectionHistory_host[i]);
		halfPopulationInfected += infectionHistory_host[i];
		allPopulationInfected += infectionHistory_host[i];
		if (infectionHistory_host[i] > 0 && !zeroDayInfectedFlag) {
			zeroDayInfected = i + 1;
			zeroDayInfectedFlag = true;
		}
		if (halfPopulationInfected >= (numberOfAgents / 2) && !halfPopulationInfectedFlag) {
			halfPopulationInfectedDay = i + 1;
			halfPopulationInfectedFlag = true;
		}
		if (allPopulationInfected == numberOfAgents)
			allPopulationInfectedDay = i + 1;
	}
	printf("\nZero day infection case: %d\n", zeroDayInfected);
	printf("Half population infected day: %d\n", halfPopulationInfectedDay);
	printf("All population infected day: %d\n", allPopulationInfectedDay);

	printf("\nTotal recovery cases: %d\n", allRecoveryCounter_host);
	printf("Recovery history: ");
	for (int i = 0; i < maxSimulationDays; i++) {
		printf("%d ", recoveryHistory_host[i]);
		halfAgentsRecovered += recoveryHistory_host[i];
		allAgentsRecovered += recoveryHistory_host[i];
		if (recoveryHistory_host[i] > 0 && !zeroDayRecoveredFlag) {
			zeroDayRecovered = i + 1;
			zeroDayRecoveredFlag = true;
		}
		if (halfAgentsRecovered >= (allRecoveryCounter_host / 2) && !halfAgentsRecoveredFlag) {
			halfAgentsRecoveredDay = i + 1;
			halfAgentsRecoveredFlag = true;
		}
		if (allAgentsRecovered == allRecoveryCounter_host)
			allAgentsRecoveredDay = i + 1;
	}
	printf("\nZero day recovery case: %d\n", zeroDayRecovered);
	printf("Half agents recovered day: %d\n", halfAgentsRecoveredDay);
	printf("All agents recovered day: %d\n", allAgentsRecoveredDay);

	printf("\nTotal fatal cases: %d\n", allFatalCounter_host);
	printf("Fatal history: ");
	for (int i = 0; i < maxSimulationDays; i++) {
		printf("%d ", fatalHistory_host[i]);
		halfAgentsFatal += fatalHistory_host[i];
		allAgentsFatal += fatalHistory_host[i];
		if (fatalHistory_host[i] > 0 && !zeroDayFatalFlag) {
			zeroDayFatal = i + 1;
			zeroDayFatalFlag = true;
		}
		if (halfAgentsFatal >= (allFatalCounter_host / 2) && !halfAgentsFatalFlag) {
			halfAgentsFatalDay = i + 1;
			halfAgentsFatalFlag = true;
		}
		if (allAgentsFatal == allFatalCounter_host)
			allAgentsFatalDay = i + 1;
	}
	printf("\nZero day fatal case: %d\n", zeroDayFatal);
	printf("Half agents fatal day: %d\n", halfAgentsFatalDay);
	printf("All agents fatal day: %d\n", allAgentsFatalDay);

	hipEventRecord(end_GPU, 0);
	hipEventSynchronize(end_GPU);
	float elapsedTime_GPU;
	hipEventElapsedTime(&elapsedTime_GPU, start_GPU, end_GPU);
	printf("\nTime GPU: %f miliseconds. \n", elapsedTime_GPU);

	hipEventDestroy(start_GPU);
	hipEventDestroy(end_GPU);

	free(infectionHistory_host);
	free(recoveryHistory_host);
	free(fatalHistory_host);

	hipFree(allAgents_dev);
	hipFree(allInfectionsCounter_dev);
	hipFree(infectionHistory_dev);
	hipFree(allRecoveryCounter_dev);
	hipFree(recoveryHistory_dev);
	hipFree(allFatalCounter_dev);
	hipFree(fatalHistory_dev);
	hipFree(counterControl_dev);
	hipFree(historyCounter_dev);

	return 0;
}