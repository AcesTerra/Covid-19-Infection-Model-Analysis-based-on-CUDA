
#include "hip/hip_runtime.h"

#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <ctime>
//#include <math.h>
#include <cmath>
#include <vector>
#include <iostream>

using namespace std;

// Agent structure with all properties
struct agent {
	float infectionProb;			// [0.02, 0.03]
	float externalInfectionProb;	// [0.02, 0.03]
	float mortalityProb;			// [0.007, 0.07]
	float mobilityProb;				// [0.3, 0.5]
	float shortMobilityProb;		// [0.7, 0.9]
	int incubationTime;				// [5, 6]
	int recoveryTime;				// 14
	int infectionStatus;			// Non infected (0), infected (1), quarantine (-1), deseaced (-2), cured (2)
	float x;						// [0, p]
	float y;						// [0, q]
};

// Simulation parameters
const int numberOfAgents = 1024;
const int maxSimulationDays = 30;
const int maxMovementsPerDay = 10;
const float maximumRadiusForLocalMovements = 5;
const float infectionLimitDistance = 1;
const float p = 500;
const float q = 500;

int allInfectionsCounter = 0;
int infectionsPerDay = 0;
int infectionHistory[maxSimulationDays];
int allRecoveryCounter = 0;
int recoveryPerDay = 0;
int recoveryHistory[maxSimulationDays];
int allFatalCounter = 0;
int fatalPerDay = 0;
int fatalHistory[maxSimulationDays];

// Function to generate a random float between a range
float generateRandom(float a, float b) {
	float r = a + static_cast <float> (rand()) / (static_cast <float> (RAND_MAX / (b - a)));
	return r;
}

// Function to move a short distance an agent
float shortMovement(float pos) {
	float newPos = (2 * generateRandom(0.0, 1.0) - 1) * maximumRadiusForLocalMovements + pos;
	if (newPos > 500)
		newPos = 500;
	if (newPos < 0)
		newPos = 0;
	return newPos;
}

// Function to move a long distance in X an agent
float longXMovement(float pos) {
	float newPos = p * generateRandom(-1.0, 1.0) + pos;
	if (newPos > 500)
		newPos = 500;
	if (newPos < 0)
		newPos = 0;
	return newPos;
}

// Function to move a long distance in Y an agent
float longYMovement(float pos) {
	float newPos = q * generateRandom(-1.0, 1.0) + pos;
	if (newPos > 500)
		newPos = 500;
	if (newPos < 0)
		newPos = 0;
	return newPos;
}

// Test function to initalize infected agents
void initializeInfectedAgents(agent allAgents[]) {
	for (int i = 0; i < numberOfAgents; i++) {
		allAgents[i].infectionProb = generateRandom(0.02, 0.03);
		allAgents[i].externalInfectionProb = generateRandom(0.02, 0.03);
		allAgents[i].mortalityProb = generateRandom(0.007, 0.07);
		allAgents[i].mobilityProb = generateRandom(0.3, 0.5);
		allAgents[i].shortMobilityProb = generateRandom(0.7, 0.9);
		allAgents[i].incubationTime = rand() % 2 + 5;
		allAgents[i].recoveryTime = 14;
		allAgents[i].infectionStatus = rand() % 2;
		allAgents[i].x = rand() % (int)p + 1;
		allAgents[i].y = rand() % (int)q + 1;
	}
}

// Function to initialize all agent properties
void initializeAgents(agent allAgents[]) {
	for (int i = 0; i < numberOfAgents; i++) {
		allAgents[i].infectionProb = generateRandom(0.02, 0.03);
		allAgents[i].externalInfectionProb = generateRandom(0.02, 0.03);
		allAgents[i].mortalityProb = generateRandom(0.007, 0.07);
		allAgents[i].mobilityProb = generateRandom(0.3, 0.5);
		allAgents[i].shortMobilityProb = generateRandom(0.7, 0.9);
		allAgents[i].incubationTime = rand() % 2 + 5;
		allAgents[i].recoveryTime = 14;
		allAgents[i].infectionStatus = 0;
		allAgents[i].x = rand() % (int)p + 1;
		allAgents[i].y = rand() % (int)q + 1;
	}
}

// Function to show all agents properties
void showAgents(agent allAgents[]) {
	for (int i = 0; i < numberOfAgents; i++) {
		printf("Agent's no. %d probability of infection: %f\n", i + 1, allAgents[i].infectionProb);
		printf("Agent's no. %d external probability of infection: %f\n", i + 1, allAgents[i].externalInfectionProb);
		printf("Agent's no. %d probability of mortality: %f\n", i + 1, allAgents[i].mortalityProb);
		printf("Agent's no. %d probability of mobility: %f\n", i + 1, allAgents[i].mobilityProb);
		printf("Agent's no. %d probability of short mobility: %f\n", i + 1, allAgents[i].shortMobilityProb);
		printf("Agent's no. %d incubation time: %d\n", i + 1, allAgents[i].incubationTime);
		printf("Agent's no. %d recovery time: %d\n", i + 1, allAgents[i].recoveryTime);
		printf("Agent's no. %d infection status: %d\n", i + 1, allAgents[i].infectionStatus);
		printf("Agent's no. %d x position: %f\n", i + 1, allAgents[i].x);
		printf("Agent's no. %d y position: %f\n\n", i + 1, allAgents[i].y);
	}
}
// Rule 1: Infection
void ruleOne(agent agents[]) {
	for (int i = 0; i < numberOfAgents; i++) {
		for (int j = 0; j < numberOfAgents; j++) {
			double distance = sqrt(pow(agents[i].x - agents[j].x, 2.0) + pow(agents[i].y - agents[j].y, 2.0));
			if (distance <= 1.0 && agents[j].infectionStatus == 1 && agents[i].infectionStatus == 0 && i != j) {
				float infection = generateRandom(0.0, 1.0);
				if (infection <= agents[i].infectionProb) {
					agents[i].infectionStatus = 1;
					allInfectionsCounter++;
					infectionsPerDay++;
				}
			}
		}
	}
}

// Rule 2: Mobility
void ruleTwo(agent agents[]) {
	for (int i = 0; i < numberOfAgents; i++) {
		float movProb = generateRandom(0.0, 1.0);
		if (movProb <= agents[i].mobilityProb && (agents[i].infectionStatus == 0 || agents[i].infectionStatus == 1)) {
			float shortMovProb = generateRandom(0.0, 1.0);
			float newXPos, newYPos;
			if (shortMovProb <= agents[i].shortMobilityProb) {
				newXPos = shortMovement(agents[i].x);
				newYPos = shortMovement(agents[i].y);
				agents[i].x = newXPos;
				agents[i].y = newYPos;
			}
			else {
				newXPos = longXMovement(agents[i].x);
				newYPos = longYMovement(agents[i].y);
				agents[i].x = newXPos;
				agents[i].y = newYPos;
			}
		}
	}
}

// Rule 3: External infection
void ruleThree(agent agents[]) {
	for (int i = 0; i < numberOfAgents; i++) {
		float infectionExternal = generateRandom(0.0, 1.0);
		if (infectionExternal <= agents[i].externalInfectionProb && agents[i].infectionStatus == 0) {
			agents[i].infectionStatus = 1;
			allInfectionsCounter++;
			infectionsPerDay++;
		}
	}
}

// Rule 4: Incucation time, symptoms, quarantine and recovery time
void ruleFour(agent agents[]) {
	for (int i = 0; i < numberOfAgents; i++) {
		if (agents[i].infectionStatus == -1 && agents[i].recoveryTime > 0) {
			agents[i].recoveryTime = agents[i].recoveryTime - 1;
		}
		if (agents[i].infectionStatus == 1 && agents[i].incubationTime > 0) {
			agents[i].incubationTime = agents[i].incubationTime - 1;
		}
		if (agents[i].infectionStatus == 1 && agents[i].incubationTime == 0) {
			agents[i].infectionStatus = -1;
		}
		if (agents[i].infectionStatus == -1 && agents[i].recoveryTime == 0) {
			agents[i].infectionStatus = 2;
			allRecoveryCounter++;
			recoveryPerDay++;
		}
	}
}

// Rule 5: Fatal cases
void ruleFive(agent agents[]) {
	for (int i = 0; i < numberOfAgents; i++) {
		float fatal = generateRandom(0.0, 1.0);
		if (fatal <= agents[i].mortalityProb && agents[i].infectionStatus == -1) {
			agents[i].infectionStatus = -2;
			allFatalCounter++;
			fatalPerDay++;
		}
	}
}

int main() {
	/*
	*************************************************************
	******************** Initalization phase ********************
	*************************************************************
	*/
	clock_t start_CPU = clock();
	srand((int)time(0));
	agent* allAgents;
	allAgents = (agent*)malloc(numberOfAgents * sizeof(agent));
	initializeAgents(allAgents);

	printf("---------------------Simulation parameters---------------------\n");
	printf("\nNumber of agents: %d\n", numberOfAgents);
	printf("Simulation days: %d\n", maxSimulationDays);
	printf("Max movements per day: %d\n", maxMovementsPerDay);
	printf("Maximum radius for local movements: %f\n", maximumRadiusForLocalMovements);
	printf("Infection limit distance: %f\n", infectionLimitDistance);
	printf("P: %f\n", p);
	printf("Q: %f\n", q);
	printf("\n--------------------Initializing simulation--------------------\n");

	/*
	*************************************************************
	********************** Operation phase **********************
	*************************************************************
	*/

	for (int day = 0; day < maxSimulationDays; day++) {
		for (int mov = 0; mov < maxMovementsPerDay; mov++) {
			ruleOne(allAgents);
			ruleTwo(allAgents);
		}
		ruleThree(allAgents);
		ruleFour(allAgents);
		ruleFive(allAgents);
		infectionHistory[day] = infectionsPerDay;
		recoveryHistory[day] = recoveryPerDay;
		fatalHistory[day] = fatalPerDay;
		infectionsPerDay = 0;
		recoveryPerDay = 0;
		fatalPerDay = 0;
	}

	/*
	*************************************************************
	************************ Show results ***********************
	*************************************************************
	*/
	printf("\n---------------------Simulation terminated---------------------\n");

	int zeroDayInfected = 0, halfPopulationInfected = 0, allPopulationInfected = 0;
	int zeroDayRecovered = 0, halfAgentsRecovered = 0, allAgentsRecovered = 0;
	int zeroDayFatal = 0, halfAgentsFatal = 0, allAgentsFatal = 0;
	int halfPopulationInfectedDay = 0, allPopulationInfectedDay = 0;
	int halfAgentsRecoveredDay = 0, allAgentsRecoveredDay = 0;
	int halfAgentsFatalDay = 0, allAgentsFatalDay = 0;
	bool zeroDayInfectedFlag = false, halfPopulationInfectedFlag = false;
	bool zeroDayRecoveredFlag = false, halfAgentsRecoveredFlag = false;
	bool zeroDayFatalFlag = false, halfAgentsFatalFlag = false;

	printf("\nTotal infected cases: %d\n", allInfectionsCounter);
	printf("Infection history: ");
	for (int i = 0; i < maxSimulationDays; i++) {
		printf("%d ", infectionHistory[i]);
		halfPopulationInfected += infectionHistory[i];
		allPopulationInfected += infectionHistory[i];
		if (infectionHistory[i] > 0 && !zeroDayInfectedFlag) {
			zeroDayInfected = i + 1;
			zeroDayInfectedFlag = true;
		}
		if (halfPopulationInfected >= (numberOfAgents / 2) && !halfPopulationInfectedFlag) {
			halfPopulationInfectedDay = i + 1;
			halfPopulationInfectedFlag = true;
		}
		if (allPopulationInfected == numberOfAgents)
			allPopulationInfectedDay = i + 1;
	}
	printf("\nZero day infection case: %d\n", zeroDayInfected);
	printf("Half population infected day: %d\n", halfPopulationInfectedDay);
	printf("All population infected day: %d\n", allPopulationInfectedDay);

	printf("\nTotal recovery cases: %d\n", allRecoveryCounter);
	printf("Recovery history: ");
	for (int i = 0; i < maxSimulationDays; i++) {
		printf("%d ", recoveryHistory[i]);
		halfAgentsRecovered += recoveryHistory[i];
		allAgentsRecovered += recoveryHistory[i];
		if (recoveryHistory[i] > 0 && !zeroDayRecoveredFlag) {
			zeroDayRecovered = i + 1;
			zeroDayRecoveredFlag = true;
		}
		if (halfAgentsRecovered >= (allRecoveryCounter / 2) && !halfAgentsRecoveredFlag) {
			halfAgentsRecoveredDay = i + 1;
			halfAgentsRecoveredFlag = true;
		}
		if (allAgentsRecovered == allRecoveryCounter)
			allAgentsRecoveredDay = i + 1;
	}
	printf("\nZero day recovery case: %d\n", zeroDayRecovered);
	printf("Half agents recovered day: %d\n", halfAgentsRecoveredDay);
	printf("All agents recovered day: %d\n", allAgentsRecoveredDay);

	printf("\nTotal fatal cases: %d\n", allFatalCounter);
	printf("Fatal history: ");
	for (int i = 0; i < maxSimulationDays; i++) {
		printf("%d ", fatalHistory[i]);
		halfAgentsFatal += fatalHistory[i];
		allAgentsFatal += fatalHistory[i];
		if (fatalHistory[i] > 0 && !zeroDayFatalFlag) {
			zeroDayFatal = i + 1;
			zeroDayFatalFlag = true;
		}
		if (halfAgentsFatal >= (allFatalCounter / 2) && !halfAgentsFatalFlag) {
			halfAgentsFatalDay = i + 1;
			halfAgentsFatalFlag = true;
		}
		if (allAgentsFatal == allFatalCounter)
			allAgentsFatalDay = i + 1;
	}
	printf("\nZero day fatal case: %d\n", zeroDayFatal);
	printf("Half agents fatal day: %d\n", halfAgentsFatalDay);
	printf("All agents fatal day: %d\n", allAgentsFatalDay);


	clock_t end_CPU = clock();
	float elapsedTime_CPU = end_CPU - start_CPU;
	printf("\nTime CPU: %f miliseconds. \n", elapsedTime_CPU);
	free(allAgents);
	return 0;
}